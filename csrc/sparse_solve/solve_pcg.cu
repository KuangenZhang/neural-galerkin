#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <time.h>
#include "err_check.cuh"

__global__ void apply_jacobi(const float *a, const float *b, float *res, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        res[i] = a[i] * b[i];
    }
}

std::pair<torch::Tensor, int> solve_pcg_diag(
        const torch::Tensor& Ap, const torch::Tensor& Aj, const torch::Tensor& Ax, const torch::Tensor& b,
        const torch::Tensor& inv_diag_A,
        const float tol, const int max_iter, const bool res_fix) {

    CHECK_CONTIGUOUS(Ap); CHECK_CUDA(Ap); CHECK_IS_INT(Ap);
    CHECK_CONTIGUOUS(Aj); CHECK_CUDA(Aj); CHECK_IS_INT(Aj);
    CHECK_CONTIGUOUS(Ax); CHECK_CUDA(Ax); CHECK_IS_FLOAT(Ax);
    CHECK_CONTIGUOUS(b); CHECK_CUDA(b); CHECK_IS_FLOAT(b);
    CHECK_CONTIGUOUS(inv_diag_A); CHECK_CUDA(inv_diag_A); CHECK_IS_FLOAT(inv_diag_A);

    int N = b.size(0);
    int nz = Ax.size(0);
    int sqrt_n = (int) std::ceil(std::sqrt((double) N));

    float b_norm = torch::linalg_norm(b).item<float>();
    float atol = tol * b_norm;

    torch::Tensor x_tensor = torch::zeros({N}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    torch::Tensor d_p_tensor = torch::zeros({N}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    torch::Tensor d_z_tensor = torch::zeros({N}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    torch::Tensor d_r_tensor = torch::clone(b);
    torch::Tensor Ax_tensor = torch::zeros({N}, torch::dtype(torch::kFloat32).device(torch::kCUDA));

    float* d_x = x_tensor.data_ptr<float>();
    float* d_p = d_p_tensor.data_ptr<float>();
    float* d_z = d_z_tensor.data_ptr<float>();
    float* d_r = d_r_tensor.data_ptr<float>();
    float* d_Ax = Ax_tensor.data_ptr<float>();
    float* d_b = b.data_ptr<float>();
    const float* d_inv_diag_A = inv_diag_A.data_ptr<float>();

    hipsparseHandle_t cusparseHandle = at::cuda::getCurrentCUDASparseHandle();
    hipblasHandle_t cublasHandle = at::cuda::getCurrentCUDABlasHandle();

    hipsparseSpMatDescr_t matA = NULL;
    cusparseSafeCall(hipsparseCreateCsr(&matA, N, N, nz, Ap.data_ptr<int>(), Aj.data_ptr<int>(), Ax.data_ptr<float>(),
                                       HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                       HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    hipsparseDnVecDescr_t vecx = NULL;
    cusparseSafeCall(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_32F));
    hipsparseDnVecDescr_t vecp = NULL;
    cusparseSafeCall(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_32F));
    hipsparseDnVecDescr_t vecAx = NULL;
    cusparseSafeCall(hipsparseCreateDnVec(&vecAx, N, d_Ax, HIP_R_32F));

    float one = 1.0;
    float neg_one = -1.0;
    float zero = 0.0;
    size_t bufferSize = 0;
    cusparseSafeCall(hipsparseSpMV_bufferSize(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecx,
            &zero, vecAx, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &bufferSize));
    torch::Tensor cusparse_buffer_tensor = torch::zeros({(int)bufferSize}, torch::dtype(torch::kByte).device(torch::kCUDA));
    void *buffer = cusparse_buffer_tensor.data_ptr<unsigned char>();

    cusparseSafeCall(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                  &one, matA, vecx, &zero, vecAx, HIP_R_32F,
                                  HIPSPARSE_MV_ALG_DEFAULT, buffer));
    cublasSafeCall(hipblasSaxpy(cublasHandle, N, &neg_one, d_Ax, 1, d_r, 1));

    int iters = 0;
    float rho = 0.0;
    float rho1;

    while (max_iter < 0 || iters < max_iter) {
        {
            dim3 dimBlock = dim3(256);
            dim3 dimGrid = dim3((N + dimBlock.x - 1) / dimBlock.x);
            apply_jacobi<<<dimGrid, dimBlock>>>(d_inv_diag_A, d_r, d_z, N);
            C10_CUDA_KERNEL_LAUNCH_CHECK();
        }

        rho1 = rho;

        cublasSafeCall(hipblasSdot(cublasHandle, N, d_r, 1, d_z, 1, &rho));

        if (iters == 0) {
            cublasSafeCall(hipblasScopy(cublasHandle, N, d_z, 1, d_p, 1));
        } else {
            float betap = rho / rho1;
            cublasSafeCall(hipblasSscal(cublasHandle, N, &betap, d_p, 1));
            cublasSafeCall(hipblasSaxpy(cublasHandle, N, &one, d_z, 1, d_p, 1));
        }

        cusparseSafeCall(hipsparseSpMV(
                cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecp,
                &zero, vecAx, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));

        float alpha, neg_alpha, dot;
        cublasSafeCall(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot));
        alpha = rho / dot;
        cublasSafeCall(hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1));

        if ((iters + 1) % sqrt_n == 0 && res_fix) {
            cusparseSafeCall(hipsparseSpMV(
                    cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, matA, vecx,
                    &zero, vecAx, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
            cublasSafeCall(hipblasScopy(cublasHandle, N, d_b, 1, d_r, 1));
            cublasSafeCall(hipblasSaxpy(cublasHandle, N, &neg_one, d_Ax, 1, d_r, 1));
        } else {
            neg_alpha = -alpha;
            cublasSafeCall(hipblasSaxpy(cublasHandle, N, &neg_alpha, d_Ax, 1, d_r, 1));
        }

        iters++;

        float resid;
        cublasSafeCall(hipblasSnrm2(cublasHandle, N, d_r, 1, &resid));
        if (resid <= atol) {
            break;
        }

    }

    cusparseSafeCall(hipsparseDestroySpMat(matA));
    cusparseSafeCall(hipsparseDestroyDnVec(vecx));
    cusparseSafeCall(hipsparseDestroyDnVec(vecAx));
    cusparseSafeCall(hipsparseDestroyDnVec(vecp));

    return std::make_pair(x_tensor, iters);
}
