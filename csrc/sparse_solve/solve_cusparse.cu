#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <hipsparse.h>
#include <hipsolver.h>
#include <time.h>
#include "err_check.cuh"

hipsolverSpHandle_t solver_handle = nullptr;

void init_cusolver_handle() {
    cusolveSafeCall(hipsolverSpCreate(&solver_handle));
}

// CuSolver only supports CSR matrix.
torch::Tensor solve_cusparse(const torch::Tensor& Ap, const torch::Tensor& Aj, const torch::Tensor& Ax, const torch::Tensor& b,
                             float tol) {
    CHECK_CONTIGUOUS(Ap); CHECK_CUDA(Ap); CHECK_IS_INT(Ap);
    CHECK_CONTIGUOUS(Aj); CHECK_CUDA(Aj); CHECK_IS_INT(Aj);
    CHECK_CONTIGUOUS(Ax); CHECK_CUDA(Ax); CHECK_IS_FLOAT(Ax);
    CHECK_CONTIGUOUS(b); CHECK_CUDA(b); CHECK_IS_FLOAT(b);

    hipsparseMatDescr_t descr;
    cusparseSafeCall(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);    // can be symmetric, triangular, etc.
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int n = b.size(0);
    int nnz = Ax.size(0);
    torch::Tensor x = torch::empty({n}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    float* x_data = x.data_ptr<float>();

    hipStreamSynchronize(at::cuda::getCurrentCUDAStream());

    int singularity;
    cusolveSafeCall(hipsolverSpScsrlsvchol(solver_handle, n, nnz, descr, Ax.data_ptr<float>(),
                    Ap.data_ptr<int>(), Aj.data_ptr<int>(), b.data_ptr<float>(),
                                          tol, 3, x_data, &singularity));

    cusparseSafeCall(hipsparseDestroyMatDescr(descr));

    return x;
}
